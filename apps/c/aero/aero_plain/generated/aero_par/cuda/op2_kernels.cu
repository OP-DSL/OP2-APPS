__constant__ double gam_d;
__constant__ double gm1_d;
__constant__ double gm1i_d;
__constant__ double m2_d;
__constant__ double wtg1_d[2];
__constant__ double xi1_d[2];
__constant__ double Ng1_d[4];
__constant__ double Ng1_xi_d[4];
__constant__ double wtg2_d[4];
__constant__ double Ng2_d[16];
__constant__ double Ng2_xi_d[32];
__constant__ double minf_d;
__constant__ double freq_d;
__constant__ double kappa_d;
__constant__ double nmode_d;
__constant__ double mfan_d;

#include "op_lib_cpp.h"
#include "op_cuda_rt_support.h"
#include "op_cuda_reduction.h"

#ifndef MAX_CONST_SIZE
#define MAX_CONST_SIZE 128
#endif

void op_decl_const_char(int dim, const char *type, int size, char *dat, const char *name) {
    if (!OP_hybrid_gpu) return;

    if (size > MAX_CONST_SIZE) {
        printf("error: requested size %d for const %s exceeds MAX_CONST_SIZE\n", size, name);
        exit(1);
    }

    if (!strcmp(name, "gam")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gam_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "gm1")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gm1_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "gm1i")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(gm1i_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "m2")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(m2_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "wtg1")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(wtg1_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "xi1")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(xi1_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "Ng1")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(Ng1_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "Ng1_xi")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(Ng1_xi_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "wtg2")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(wtg2_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "Ng2")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(Ng2_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "Ng2_xi")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(Ng2_xi_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "minf")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(minf_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "freq")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(freq_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "kappa")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(kappa_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "nmode")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(nmode_d), dat, dim * size));
        return;
    }
    if (!strcmp(name, "mfan")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(mfan_d), dat, dim * size));
        return;
    }

    printf("error: unknown const name %s\n", name);
    exit(1);
}

#include "aero_mpi_1_res_calc_kernel.hpp"
#include "aero_mpi_2_dirichlet_kernel.hpp"
#include "aero_mpi_3_init_cg_kernel.hpp"
#include "aero_mpi_4_spMV_kernel.hpp"
#include "aero_mpi_5_dirichlet_kernel.hpp"
#include "aero_mpi_6_dotPV_kernel.hpp"
#include "aero_mpi_7_updateUR_kernel.hpp"
#include "aero_mpi_8_dotR_kernel.hpp"
#include "aero_mpi_9_updateP_kernel.hpp"
#include "aero_mpi_10_update_kernel.hpp"
