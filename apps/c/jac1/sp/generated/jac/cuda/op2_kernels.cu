__constant__ float alpha_d;

#include "op_lib_cpp.h"
#include "op_cuda_rt_support.h"
#include "op_cuda_reduction.h"

#ifndef MAX_CONST_SIZE
#define MAX_CONST_SIZE 128
#endif

void op_decl_const_char(int dim, const char *type, int size, char *dat, const char *name) {
    if (!OP_hybrid_gpu) return;

    if (size > MAX_CONST_SIZE) {
        printf("error: requested size %d for const %s exceeds MAX_CONST_SIZE\n", size, name);
        exit(1);
    }

    if (!strcmp(name, "alpha")) {
        cutilSafeCall(hipMemcpyToSymbol(HIP_SYMBOL(alpha_d), dat, dim * size));
        return;
    }

    printf("error: unknown const name %s\n", name);
    exit(1);
}

#include "jac_1_res_kernel.hpp"
#include "jac_2_update_kernel.hpp"
